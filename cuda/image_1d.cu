#include "hip/hip_runtime.h"
#include "image.hpp"

/**
 * @brief kernel that squares each element of array
 * 
 * @param arr array pointer
 * @param arrSize array size
 */
__global__ void squareArray(float* arr, int arrSize)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < arrSize)
    {
        const float tmp = arr[i];
        arr[i] = tmp * tmp;
    }
}

// Image initialization 
const int width = 1024;
const int height = 1024;
using Image2D = Image<float, width, height>;
Image2D hostImage;

int main()
{
    // Fill image 
    for (int r =0; r < height; ++r)
    {
        for (int c =0; c < width; ++c)
        {
            hostImage.data[r*width+c] = (r*width+c)/1024.f;
        }
    }

    // Allocate memory for image on device
    Image2D::value_type* deviceImage;
    hipMalloc(&deviceImage, width*height*sizeof(Image2D::value_type));

    // Copy memory from host memory to device memory
    hipMemcpy(deviceImage, hostImage.data, width*height*sizeof(Image2D::value_type), hipMemcpyHostToDevice);

    // Invoke kernel
    int threadsPerblock = 256;
    int blocksPerGrid = ( width*height + threadsPerblock  - 1) / threadsPerblock;
    squareArray<<<blocksPerGrid, threadsPerblock>>>(deviceImage, width*height);

    // Copy memory from device memory to host memory
    hipMemcpy(hostImage.data, deviceImage, width*height*sizeof(Image2D::value_type), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(deviceImage);

    return 0;
}