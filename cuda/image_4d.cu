#include "hip/hip_runtime.h"
#include "tensor.hpp"

/**
 * @brief kernel that squares each element of array representing 4D image (bxwxhxc)
 * 
 * @param arr array pointer
 * @param arrSize array size
 */
__global__ void square2DMatrix(float* matrix, int width, int height, int channels, int batchSize)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    // Number of threads in z dim is equal to batchSize*channels
    int ch = (blockDim.z * blockIdx.z + threadIdx.z) / batchSize;
    int batch = (blockDim.z * blockIdx.z + threadIdx.z) % batchSize;  

    if (col < width && row < height && ch < channels && batch < batchSize)
    {
        int index = batch*channels*height*wight + ch*row*width + row*width + col;
        matrix[index] *= matrix[index];
    }
}

// Image initialization 
const int width = 1024;
const int height = 1024;
const int channels = 3;
const int batches = 4;
using Image4D = Tenosr<float, width, height, channels, batches>;
Image4D hostImage;

int main()
{
    // Fill image 
    for (int b=0; b < batches; ++b)
    {
        for (int ch=0; ch < channels; ++ch)
        {
            for (int r =0; r < height; ++r)
            {
                for (int c =0; c < width; ++c)
                {
                    hostImage.data[b*channels*width*height + ch*width*height + r*width + c] = 1/1024.f;
                }
            }
        }
    }

    // Allocate memory for image on device
    Image4D::value_type* deviceImage;
    hipMalloc(&deviceImage, width*height*channels*batches*sizeof(Image4D::value_type));

    // Copy memory from host memory to device memory
    hipMemcpy(deviceImage, hostImage.data, width*height*channels*batches*sizeof(Image4D::value_type), hipMemcpyHostToDevice);

    1024x1024x3
    widthxheightxchannel
    batchxheightxwidthxchannel

    // Invoke kernel
    dim3 threadsPerblock(16, 16, 1);
    dim3 blocksPerGrid( (width + threadsPerblock.x - 1) / threadsPerblock.x, 
    (height + threadsPerblock.y - 1) / threadsPerblock.y, (channels * batches + threadsPerblock.z - 1) / threadsPerblock.z);
    square2DMatrix<<<blocksPerGrid, threadsPerblock>>>(deviceImage, width, height, channels, batches);

    // Copy memory from device memory to host memory 
    hipMemcpy(hostImage.data, deviceImage, width*height*channels*batches*sizeof(Image4D::value_type), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(deviceImage);

    return 0;
}