#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>
#include <cmath>

#include "matrix.hpp"

/**
 * @brief kernel that performs matrix multiplication, C = matMul(A, B), in parallel
 * 
 * @param A array pointer of 2D matrix A
 * @param B array pointer of 2D matrix B
 * @param C array pointer of 2D matrix C
 * @param w_a width of matrix A
 * @param w_c width of matix C
 * @param h_c height of matrix C
 */
__global__ void matMulKernel(float* A, float* B, float* C, int w_a, int w_c, int h_c)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    if (col < w_c && row < h_c)
    {
        for (int i = 0; i < w_a; ++i)
        {
            C[row*w_c+col] += ( A[row*w_a+i] * B[i*h_c + col] );
        }
    }
}

/**
 * @brief Kernel performing elementwise sigmoid
 * 
 * @param A matrix A
 * @param w width of matrix 
 * @param h height of matrix
 */
__global__ void sigmoidKernel(float* A, int w, int h)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    if (col < w && row < h)
    {
        int index = row*w + col;
        A[index] = 1.0f / (1.0f + expf(-A[index]));
    }
}


/**
 * @brief performs matrix multiplication and elementwise sigmoid
 * 
 * @param A  matrix A
 * @param B  matrix B
 * @param C  matrix C
 */
void matrixOperations(const Matrix& A, const Matrix& B, Matrix& C)
{
    // Allocate memory for matrix A, b and C on device
    float* A_d;
    float* B_d;
    float* C_d;
    hipMalloc(&A_d, A.width*A.height*sizeof(float));
    hipMalloc(&B_d, B.width*B.height*sizeof(float));
    hipMalloc(&C_d, C.width*C.height*sizeof(float));
    hipMemset(C_d, 0.0f, C.width*C.height*sizeof(float));

    // Copy data from host memory to device memory
    hipMemcpy(A_d, A.data.get(), A.width*A.height*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B.data.get(), B.width*B.height*sizeof(float), hipMemcpyHostToDevice);

    // Invoke kernels
    dim3 threadsPerblock(32, 16);
    dim3 blocksPerGrid( (C.width + threadsPerblock.x - 1) / threadsPerblock.x, 
    (C.height + threadsPerblock.y - 1) / threadsPerblock.y );

    // Launch matMulKernel 
    matMulKernel<<<blocksPerGrid, threadsPerblock>>>(A_d, B_d, C_d, A.width, C.width, C.height);

    // Launch sigmoidKernel
    sigmoidKernel<<<blocksPerGrid, threadsPerblock>>>(C_d, C.width, C.height);

    // Copy memory from device memory to host memory
    hipMemcpy(C.data.get(), C_d, C.width*C.height*sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

/**
 * @brief performs matrix multiplication C = matMul(A, B) and elementwise sigmoid
 * 
 * @param A matrix A
 * @param B matrix B
 * @return Matrix C
 */
Matrix matrixOperations(const Matrix& A, const Matrix& B)
{
    if (A.width != B.height )
    {
        return Matrix();
    }
    // Initialize matrix C
    Matrix C(B.width, A.height);
    matrixOperations(A, B, C);
    return C;
}

/**
 * @brief Helper function to check equality within two decimal precision
 * 
 * @param a first float value
 * @param b second float value
 * @param precision precision 
 * @return true if precision met
 */
bool isApproxEqual(float a, float b, float precision = 0.01f) {
    return std::fabs(a - b) < precision;
}

int main()
{
    // Initialize matrix A 
    Matrix A(3, 4);
    A.data.get()[0] = 0.1f;
    A.data.get()[1] = 0.2f;
    A.data.get()[2] = 0.3f;
    A.data.get()[3] = 0.4f;
    A.data.get()[4] = 0.5f;
    A.data.get()[5] = 0.6f;
    A.data.get()[6] = 0.7f;
    A.data.get()[7] = 0.8f;
    A.data.get()[8] = 0.9f;
    A.data.get()[9] = 0.10f;
    A.data.get()[10] = 0.11f;
    A.data.get()[11] = 0.12f;

    // Initialize matrix B
    Matrix B(2, 3);
    B.data.get()[0] = 0.1f;
    B.data.get()[1] = 0.2f;
    B.data.get()[2] = 0.3f;
    B.data.get()[3] = 0.4f;
    B.data.get()[4] = 0.5f;
    B.data.get()[5] = 0.6f;

    // Matrix multiplication
    auto C = matrixOperations(A, B);

    // Check the results
    assert( isApproxEqual(C.data.get()[0], 0.55f) );
    assert( isApproxEqual(C.data.get()[1], 0.56f) );
    assert( isApproxEqual(C.data.get()[2], 0.62f) );
    assert( isApproxEqual(C.data.get()[3], 0.65f) );
    assert( isApproxEqual(C.data.get()[4], 0.68f) );
    assert( isApproxEqual(C.data.get()[5], 0.73f) );
    assert( isApproxEqual(C.data.get()[6], 0.52f) );
    assert( isApproxEqual(C.data.get()[7], 0.53f) );

    std::cout << "All tests passed!" << std::endl;

    return 0;
}