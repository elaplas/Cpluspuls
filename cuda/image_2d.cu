#include "hip/hip_runtime.h"
#include "image.hpp"

/**
 * @brief kernel that squares each element of array
 * 
 * @param arr array pointer
 * @param arrSize array size
 */
__global__ void square2DMatrix(float* matrix, int width, int height)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    if (col < width && row < height)
    {
        int index = row*width+col;
        matrix[index] *= matrix[index];
    }
}

// Image initialization 
const int width = 1024;
const int height = 1024;
using Image2D = Image<float, width, height>;
Image2D hostImage;

int main()
{
    // Fill image 
    for (int r =0; r < height; ++r)
    {
        for (int c =0; c < width; ++c)
        {
            hostImage.data[r*width+c] = (r*width+c)/1024.f;
        }
    }

    // Allocate memory for image on device
    Image2D::value_type* deviceImage;
    hipMalloc(&deviceImage, width*height*sizeof(Image2D::value_type));

    // Copy memory from host memory to device memory
    hipMemcpy(deviceImage, hostImage.data, width*height*sizeof(Image2D::value_type), hipMemcpyHostToDevice);

    // Invoke kernel
    dim3 threadsPerblock(32, 16);
    dim3 blocksPerGrid( (width + threadsPerblock.x - 1) / threadsPerblock.x, 
    (height + threadsPerblock.y - 1) / threadsPerblock.y );
    square2DMatrix<<<blocksPerGrid, threadsPerblock>>>(deviceImage, width, height);

    // Copy memory from device memory to host memory
    hipMemcpy(hostImage.data, deviceImage, width*height*sizeof(Image2D::value_type), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(deviceImage);

    return 0;
}